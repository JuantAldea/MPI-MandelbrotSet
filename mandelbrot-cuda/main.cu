#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: 22/04/2012
 *      Author: Juan Antonio Aldea Armenteros
 */

#include <hip/hip_runtime.h>
#include <stdio.h>

#define pix_per_thread 1
#define DEBUG

extern "C" {
#include "ppm.h"
}

__global__ void render(unsigned char *out, int width, int height) {
	//for (int i = 0; i < pix_per_thread; i++){
	//for (int j = 0; j < pix_per_thread; j++){

	//unsigned int x_dim = blockIdx.x * blockDim.x * pix_per_thread + threadIdx.x * pix_per_thread + i;
	//unsigned int y_dim = blockIdx.y * blockDim.y * pix_per_thread + threadIdx.y * pix_per_thread + j;
	unsigned int x_dim = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y_dim = blockIdx.y * blockDim.y + threadIdx.y;
	int index = 3 * width * y_dim + x_dim * 3;
	float x_origin = ((float) x_dim / width) * 3.25 - 2;
	float y_origin = ((float) y_dim / height) * 2.5 - 1.25;
	float x = 0.0;
	float y = 0.0;

	int iteration = 0;
	int max_iteration = 10000;

	while (x * x + y * y <= 4 && iteration < max_iteration) {
		float xtemp = x * x - y * y + x_origin;
		y = 2 * x * y + y_origin;
		x = xtemp;
		iteration++;
	}
	//out[index]++;
	if (iteration == max_iteration) {
		out[index + 0] = 0;
		out[index + 1] = 0;
		out[index + 2] = 0;
	} else {
		out[index + 0] = iteration < 255 ? iteration : 255;
		out[index + 1] = iteration < 255 ? iteration : 255;
		out[index + 2] = iteration < 255 ? iteration : 255;
	}
//    }
	//}
}

void runCUDA(int width, int height) {
	size_t buffer_size = sizeof(unsigned char) * width * height * 3;
	unsigned char *device_memory, *host_memory;
	dim3 blockDim(16, 16, 1);
	//dim3 gridDim(width / (pix_per_thread * blockDim.x), height / (2 * pix_per_thread * blockDim.y), 1);
	dim3 gridDim(width / (blockDim.x), height / (blockDim.y), 1);
	hipError_t cuda_error;
	hipDeviceReset();
	cuda_error = hipSetDeviceFlags(hipDeviceMapHost);
	printf("Set device: %s\n", hipGetErrorString(cuda_error));

	int host_alloc = 1;

	if (cuda_error == hipSuccess && host_alloc) {
		cuda_error = hipHostAlloc((void**) &host_memory, buffer_size,
				hipHostMallocMapped);
		printf("Host1 %s\n", hipGetErrorString(cuda_error));
		cuda_error = hipHostGetDevicePointer(&device_memory, host_memory, 0);
		printf("Host2 %s\n", hipGetErrorString(cuda_error));
#ifdef DEBUG
		cuda_error = hipMemset(device_memory, 255, buffer_size);
		printf("Host3 %s\n", hipGetErrorString(cuda_error));
#endif
	} else {
		cuda_error = hipMalloc((void **) &device_memory, buffer_size);
		printf("Device %s\n", hipGetErrorString(cuda_error));
#ifdef DEBUG
		cuda_error = hipMemset(device_memory, 255, buffer_size);
		printf("Device %s\n", hipGetErrorString(cuda_error));
#endif
		host_memory = (unsigned char *) malloc(buffer_size);
	}

	//ejecuta el kernel
	render<<< gridDim, blockDim, 0 >>>(device_memory, width, height);

	if (cuda_error == hipSuccess && host_alloc) {
		cuda_error = hipDeviceSynchronize();
		printf("Host barrier %s\n", hipGetErrorString(cuda_error));
	} else {
		cuda_error = hipMemcpy(host_memory, device_memory, buffer_size,
				hipMemcpyDeviceToHost);
		printf("Device %s\n", hipGetErrorString(cuda_error));
	}

	write_ppm("broza.ppm", height, width, 255, host_memory);

	if (cuda_error == hipSuccess && host_alloc) {
		cuda_error = hipHostFree(host_memory);
		printf("Host %s\n", hipGetErrorString(cuda_error));
	} else {
		cuda_error = hipFree(device_memory);
		printf("%s\n", hipGetErrorString(cuda_error));
		free(host_memory);
	}
}

int main(int argc, const char * argv[]) {
	int dim = 64;
	runCUDA(dim, dim);
	return 0;
}
